#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include <time.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>

#define THREAD_PER_BLOCK 16

void simple_moving_Average(const int array_size, const float* datapoints_array, const int N, float* output_array)
{
    int i, j;
    float sum;
    for (i = 0; i < N - 1; i++) {
        output_array[i] = datapoints_array[i];
        //printf("Simple Moving Average : CPU : %f \n", output_array[i]);
    }
    for (; i < array_size; i++) {
        sum = 0;
        for (j = 0; j < N; j++) {
            sum += datapoints_array[i - j];
        }
        output_array[i] = sum / N;
        //printf("Simple Moving Average : CPU : %f \n", output_array[i]);
    }
}

__global__ void simple_moving_Average_kernel(const float array_size, const float* datapoints_array, const int N, float* output_array)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x, j;
    float sum;
    if (i < N - 1) {
        output_array[i] = datapoints_array[i];
        //printf("Simple Moving Average : GPU: %f \n", output_array[i]);
    }
    else if (i < array_size) {
        sum = 0;
        for (j = 0; j < N; j++) {
            sum += datapoints_array[i - j];
        }
        output_array[i] = sum / N;
        //printf("Simple Moving Average : GPU: %f \n", output_array[i]);
    }
}

void simple_moving_Average_gpu(const int array_size, const float* datapoints_array, const int N, float* output_array)
{
    float* datapoints_array_gpu, * output_array_gpu;
    hipMalloc((void**)&datapoints_array_gpu, array_size * sizeof(float));
    hipMalloc((void**)&output_array_gpu, array_size * sizeof(float));
    hipMemcpy(datapoints_array_gpu, datapoints_array, array_size * sizeof(float), hipMemcpyHostToDevice);

    //Simple Moving Average GPU
    clock_t start_gpu, end_gpu;
    float total_time;
    start_gpu = clock();
    dim3 blocks(THREAD_PER_BLOCK);
    dim3 grids(array_size / THREAD_PER_BLOCK + 1);
    simple_moving_Average_kernel << <grids, blocks >> > (array_size, datapoints_array_gpu, N, output_array_gpu);
    hipMemcpy((void*)output_array, (void*)output_array_gpu, array_size * sizeof(float), hipMemcpyDeviceToHost);
    end_gpu = clock();
    //time count stops 
    total_time = ((float)(end_gpu - start_gpu)) / CLOCKS_PER_SEC;
    //calulate total time
    printf("\nTime taken to calculate moving average for GPU: %f \n", total_time);

    hipFree(datapoints_array_gpu);
    hipFree(output_array_gpu);
}

int main()
{
    const int input_array_size = 10000;
    float* sample_array = new float[input_array_size];
    float* output = new float[input_array_size];
    float total_time;
    int N;

    printf("Enter N value:\n");
    scanf("%d", &N);

    //Simple Moving Average CPU
    clock_t start, end;
    start = clock();
    
    for (int i = 0; i < input_array_size; ++i) {
        sample_array[i] = rand() % 100;
    }

    simple_moving_Average(input_array_size, sample_array, N, output);
    end = clock();
    //time count stops 
    total_time = ((float)(end - start)) / CLOCKS_PER_SEC;
    //calulate total time
    printf("\nTime taken to calculate moving average for CPU: %f\n", total_time);

    //Simple Moving Average GPU
    simple_moving_Average_gpu(input_array_size, sample_array, N, output);
   

    return 0;
}

